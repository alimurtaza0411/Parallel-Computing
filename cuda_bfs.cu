#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <fstream>
#define num_threads 1024
using namespace std;

__global__ void level_bfs(int * que , int que_size , int *next_que , int *next_que_size , int *distance , int * ad_siz , int* edges ,int * startpos )
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid<que_size)
    {
        int v = que[tid];
        for(int i = startpos[v] ; i <= startpos[v] + ad_siz[v] ; i++ )
        {
            if(atomicCAS(&(distance[i]) , -1 , distance[v] + 1) == -1)
            {
                int pos = atomicAdd(next_que_size , 1);
                next_que[pos] = i;
            }
        }
    }
}

int main(int argc, char *argv[])
{
    
    ifstream input(argv[1]);

    int num_vertices , num_edges;
    
    input>>num_vertices;

    input>>num_edges;

    int *edges = (int*)malloc(num_edges*sizeof(int));
    int *startpos = (int*)malloc(num_vertices*sizeof(int));
    int *ad_siz = (int*)malloc(num_vertices*sizeof(int));
    int *dist = (int*)malloc(num_vertices*sizeof(int));
    int *que = (int*)malloc(num_vertices*sizeof(int));
    int *que_size =(int*)malloc(sizeof(int));
    int *next_que_size =(int*)malloc(sizeof(int));

    for(int i=0;i<num_edges;i++)
    {
        input>>edges[i];
    }


    for(int i =0; i<num_vertices ; i++)
    {
        input>>startpos[i];
    }


    for(int i =0; i<num_vertices ; i++)
    {
        input>>ad_siz[i];
    }

    // memset(dist , -1 , sizeof(dist));

    for(int i = 0 ; i < num_vertices ; i++)
    dist[i] = -1;

    dist[0] = 0;
    que[0] = 0;
    *que_size = 1;
    *next_que_size = 0;


    int *d_dist , *d_edges ,*d_start_pos , *d_ad_siz , *d_que, *d_next_que , *d_que_size , *d_next_que_size;
    
    hipMalloc((void**)&d_dist , num_vertices*sizeof(int) );
    hipMalloc((void**)&d_start_pos , num_vertices*sizeof(int) );
    hipMalloc((void**)&d_ad_siz , num_vertices*sizeof(int) );
    hipMalloc((void**)&d_que , num_vertices*sizeof(int) );
    hipMalloc((void**)&d_next_que , num_vertices*sizeof(int) );
    hipMalloc((void**)&d_edges , num_edges*sizeof(int) ); 
    hipMalloc((void**)&d_que_size , sizeof(int) );
    hipMalloc((void**)&d_next_que_size , sizeof(int) );

    hipMemcpy(d_dist , dist , num_vertices*sizeof(int) , hipMemcpyHostToDevice );
    hipMemcpy(d_start_pos , startpos , num_vertices*sizeof(int) , hipMemcpyHostToDevice );
    hipMemcpy(d_ad_siz , ad_siz , num_vertices*sizeof(int) , hipMemcpyHostToDevice );
    hipMemcpy(d_que , que , num_vertices*sizeof(int) , hipMemcpyHostToDevice );
    hipMemcpy(d_edges , edges , num_vertices*sizeof(int) , hipMemcpyHostToDevice );
    hipMemcpy(d_que_size , que_size , sizeof(int) , hipMemcpyHostToDevice );
    hipMemcpy(d_next_que_size , next_que_size , sizeof(int) , hipMemcpyHostToDevice );
    
    // cout<<"HI"<<endl;

    while(*que_size>0)
    {
        long num_blocks = (*que_size+num_threads-1)/num_threads;
        // cout<<"HI"<<endl;
        level_bfs<<<num_blocks , num_threads >>>(d_que , *que_size , d_next_que ,  d_next_que_size , d_dist , d_ad_siz , d_edges , d_start_pos);

        // cout<<"HI"<<endl;
        // break;
        hipMemcpy( que_size , d_next_que_size , sizeof(int) , hipMemcpyDeviceToHost  );
        hipMemcpy( d_next_que_size , next_que_size , sizeof(int) , hipMemcpyHostToDevice );
        // cout<<*que_size<<endl;
        swap(d_next_que , d_que);

    }

    hipMemcpy( dist , d_dist , num_vertices*sizeof(int) , hipMemcpyDeviceToHost );

    for(int i = 0 ; i < num_vertices ; i++)
    {
        cout<<i<<": "<<dist[i]<<endl;
    }

}